/*
Adapted from https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf.
The idea is to do a reduction on an N-element tensor by running an
(lowPow2 / 128) x N grid of blocks, each containing 128 threads.
*/
#include <hip/hip_runtime.h>


constexpr size_t threads = 1024;

__device__ void warpReduce(volatile float* sdata, const size_t tid)
{
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid +  8];
    sdata[tid] += sdata[tid +  4];
    sdata[tid] += sdata[tid +  2];
    sdata[tid] += sdata[tid +  1];
}

__global__ void reduceAddKernel(
    const size_t batchSize,
    const size_t stride,
    const float* inp,
    float* out)
{
    __shared__ float sdata[threads];

    const size_t offset = blockIdx.y;
    const size_t tid = threadIdx.x;
    const size_t myId = 2 * blockDim.x * blockIdx.x + tid;

    sdata[tid] = myId < batchSize ? inp[offset + stride * myId] : 0;

    const size_t myId2 = myId + blockDim.x;
    if (myId2 < batchSize)
        sdata[tid] += inp[offset + stride * myId2];

    __syncthreads();

    for (size_t s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s)
            sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid < 32)
        warpReduce(sdata, tid);

    if (tid == 0)
        atomicAdd(&out[offset], sdata[0]);
}

extern "C" void reduceAdd(
    const size_t batchSize,
    const size_t tensorSize,
    const float* inp,
    float* out)
{
    const size_t dual = 2 * threads;
    const size_t grid_x = (batchSize + dual - 1) / dual;
    const dim3 grid(grid_x, tensorSize);

    reduceAddKernel<<<grid, threads>>>(
        batchSize,
        tensorSize,
        inp,
        out
    );
}
