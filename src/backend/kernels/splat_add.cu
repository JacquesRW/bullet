#include <hip/hip_runtime.h>


constexpr size_t threads = 1024;

__global__ void splatAddKernel(
    const size_t batchSize,
    const size_t stride,
    const float* inp_a,
    const float* inp_b,
    float* out)
{
    const size_t offset = blockIdx.y;
    const size_t tid = threadIdx.x;
    const size_t myId = blockDim.x * blockIdx.x + tid;

    if (myId >= batchSize)
        return;

    const size_t idx = offset + stride * myId;

    out[idx] = inp_a[offset] + inp_b[idx];
}

extern "C" void splatAdd(
    const size_t batchSize,
    const size_t tensorSize,
    const float* inp_a,
    const float* inp_b,
    float* out)
{
    const size_t grid_x = (batchSize + threads - 1) / threads;
    const dim3 grid(grid_x, tensorSize);

    splatAddKernel<<<grid, threads>>>(
        batchSize,
        tensorSize,
        inp_a,
        inp_b,
        out
    );
}
