#include "hip/hip_runtime.h"
/*
Computes
N = len(input_vector)
output_vector = input_vector[:N] * input_vector[N:]
(and gradients thereof)
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// This file is commented to death, because it was written by someone who doesn't know CUDA very well (cosmo).

constexpr size_t threadsPerBlock = static_cast<size_t>(1024);

__global__ void pairwiseMulKernel(
    const size_t batchSize,
    const size_t tensorSize,
    const float* inp,
    float* out) {
    const size_t tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= tensorSize)
        return;

    // input vector is 2x size   vvvvvvvvvvvvvv
    const float* thisInp = inp + 2 * tensorSize * blockIdx.y + tid;
    float* thisOut = out + tensorSize * blockIdx.y + tid;

    thisOut[0] = thisInp[0] * thisInp[tensorSize];
}

extern "C" void pairwiseMul(
    const size_t batchSize,
    const size_t inputSize,
    const size_t outputSize,
    const float* input,
    float* output) {
    const size_t grid_x = (outputSize + threadsPerBlock - 1) / threadsPerBlock;
    const dim3 grid(grid_x, batchSize);

    pairwiseMulKernel<<<grid, threadsPerBlock>>>(
        batchSize,
        outputSize,
        inp,
        out);
}

__global__ void pairwiseMulBackwardKernel(
    const size_t batchSize,
    const size_t tensorSize,
    const float* inp,
    float* out) {
    const size_t tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= tensorSize)
        return;

    const float* thisInp = inp + tensorSize * blockIdx.y + tid;
    float* thisOut = out + 2 * tensorSize * blockIdx.y + tid;

    // thisOut[0] = thisInp[0] * thisInp[tensorSize];
    const float gradIn = thisInp[0];
    const float valLeft = thisOut[0];
    const float valRight = thisOut[tensorSize];
    const float gradLeft = gradIn * valRight;
    const float gradRight = gradIn * valLeft;

    thisOut[0] = gradLeft;
    thisOut[tensorSize] = gradRight;
}

extern "C" void backpropPairwiseMul(
    const size_t batchSize,
    const size_t inputSize,
    const size_t outputSize,
    // gradients on the output
    const float* input,
    // buffer to write gradients into
    float* output) {
    const size_t grid_x = (inputSize + threadsPerBlock - 1) / threadsPerBlock;
    const dim3 grid(grid_x, batchSize);

    // Launch the kernel
    pairwiseMulBackwardKernel<<<grid, threadsPerBlock>>>(
        batchSize, inputSize, input, output);
}