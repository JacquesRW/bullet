#pragma once
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdint>

__global__ void addInternal(const float* A, const float* B, float* C, int size)
{
    int i = threadIdx.x;

    if (i >= size)
        return;

    C[i] = A[i] + B[i];
}

__global__ void populateAccumulator(
    const size_t batchSize,
    const size_t hiddenSize,
    const size_t inputSize,
    const float* featureWeights,
    const float* featureBiases,
    const uint16_t* inputs,
    float* accumulators)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= hiddenSize)
        return;

    const size_t inputIdx = inputSize * blockIdx.x;
    const size_t element = threadIdx.x;
    const size_t outputIdx = hiddenSize * blockIdx.x + element;

    const uint16_t* thisInput = inputs + inputIdx;

    float elementVal = featureBiases[element];

    for (size_t i = 0; i < inputSize; i++) {
        if (thisInput[i] >= static_cast<uint16_t>(768))
            break;

        const size_t idx = static_cast<size_t>(thisInput[i]) * hiddenSize + element;
        elementVal += featureWeights[idx];
    }

    if (elementVal < 0)
        elementVal = 0;
    else if (elementVal > 1)
        elementVal = 1;

    accumulators[outputIdx] = elementVal;
}

__global__ void setOutputBias(
    const size_t batchSize,
    const float* outputBias,
    float* outputs)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= batchSize)
        return;

    outputs[idx] = outputBias[0];
}

__global__ void calculateEvals(
    const size_t batchSize,
    const size_t hiddenSize,
    const float* outputWeights,
    const float* outputBiases,
    const float* ourAccumulators,
    const float* oppAccumulators,
    float* outputs)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= hiddenSize)
        return;

    const size_t element = threadIdx.x;
    const size_t outputIdx = blockIdx.x;
    const size_t idx = outputIdx * hiddenSize + element;

    float outputVal = ourAccumulators[idx] * outputWeights[element];
    outputVal += oppAccumulators[idx] * outputWeights[hiddenSize + element];

    atomicAdd(&outputs[outputIdx], outputVal);
}

__global__ void calculateErrors(
    const size_t batchSize,
    const size_t hiddenSize,
    const float* results,
    float* outputs,
    float* error)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= batchSize)
        return;

    const float eval = outputs[idx];
    const float result = results[idx];
    const float sigmoid = 1.0 / (1.0 + expf(-eval));
    const float diff = sigmoid - result;
    const float singleError = diff * sigmoid * (1.0 - sigmoid);

    atomicAdd(error, diff * diff);

    outputs[idx] = singleError;
}

__global__ void backpropSide(
    const size_t batchSize,
    const size_t hiddenSize,
    const size_t inputSize,
    const size_t outputOffset,
    const float* outputWeights,
    const float* accumulator,
    const uint16_t* inputs,
    const float* outputs,
    float* featureWeightsGradient,
    float* featureBiasesGradient,
    float* outputWeightsGradient)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= hiddenSize)
        return;

    const size_t element = threadIdx.x;
    const size_t outputIdx = blockIdx.x;
    const size_t inputIdx = outputIdx * inputSize;
    const size_t outputWeightIdx = element + outputOffset;
    const size_t accumulatorIdx = outputIdx * hiddenSize + element;

    const uint16_t* thisInput = inputs + inputIdx;

    const float error = outputs[outputIdx];
    const float weight = outputWeights[outputWeightIdx];
    const float accumulatorVal = accumulator[accumulatorIdx];

    // uses a trick
    const float component = accumulatorVal > 0 && accumulatorVal < 1
        ? error * weight
        : 0;

    atomicAdd(&featureBiasesGradient[element], component);
    atomicAdd(&outputWeightsGradient[outputWeightIdx], error * accumulatorVal);

    for (int i = 0; i < inputSize; i++) {
        if (thisInput[i] >= static_cast<uint16_t>(768))
            break;

        const size_t x = thisInput[i] * hiddenSize + element;
        atomicAdd(&featureWeightsGradient[x], component);
    }
}

__global__ void backpropOutputBias(
    const size_t batchSize,
    const float* outputs,
    float* outputBiasesGradient)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= batchSize)
        return;

    atomicAdd(outputBiasesGradient, outputs[idx]);
}

void checkError(std::string message)
{
    const auto error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cout << message << std::endl;
        std::cout << hipGetErrorString(error) << std::endl;
    }
}

extern "C" {
    hipError_t add(const float* A, const float* B, float* C, int size)
    {
        addInternal<<<1, 3>>>(A, B, C, size);

        return hipGetLastError();
    }

    hipError_t trainBatch(
        const size_t batchSize,
        const size_t hiddenSize,
        const size_t inputSize,
        const float* featureWeights,
        const float* featureBiases,
        const float* outputWeights,
        const float* outputBiases,
        const uint16_t* ourInputs,
        const uint16_t* oppInputs,
        const float* results,
        float* featureWeightsGradient,
        float* featureBiasesGradient,
        float* outputWeightsGradient,
        float* outputBiasesGradient,
        float* error)
    {
        const size_t accumulatorSize = batchSize * hiddenSize * sizeof(float);
        const size_t outputSize = batchSize * sizeof(float);
        const size_t blocks = (batchSize + hiddenSize - 1) / hiddenSize;

        float* ourAccumulators;
        hipMallocManaged(&ourAccumulators, accumulatorSize);
        hipDeviceSynchronize();

        populateAccumulator<<<batchSize, hiddenSize>>>(batchSize, hiddenSize, inputSize, featureWeights, featureBiases, ourInputs, ourAccumulators);
        hipDeviceSynchronize();

        float* oppAccumulators;
        hipMallocManaged(&oppAccumulators, accumulatorSize);
        hipDeviceSynchronize();

        populateAccumulator<<<batchSize, hiddenSize>>>(batchSize, hiddenSize, inputSize, featureWeights, featureBiases, oppInputs, oppAccumulators);
        hipDeviceSynchronize();

        float* outputs;
        hipMallocManaged(&outputs, outputSize);
        hipDeviceSynchronize();

        setOutputBias<<<blocks, hiddenSize>>>(batchSize, outputBiases, outputs);
        hipDeviceSynchronize();

        calculateEvals<<<batchSize, hiddenSize>>>(batchSize, hiddenSize, outputWeights, outputBiases, ourAccumulators, oppAccumulators, outputs);
        hipDeviceSynchronize();

        calculateErrors<<<blocks, hiddenSize>>>(batchSize, hiddenSize, results, outputs, error);
        hipDeviceSynchronize();

        backpropSide<<<batchSize, hiddenSize>>>(
            batchSize, hiddenSize, inputSize, 0,
            outputWeights, ourAccumulators, ourInputs, outputs,
            featureWeightsGradient, featureBiasesGradient, outputWeightsGradient
        );
        hipDeviceSynchronize();

        backpropSide<<<batchSize, hiddenSize>>>(
            batchSize, hiddenSize, inputSize, hiddenSize,
            outputWeights, oppAccumulators, oppInputs, outputs,
            featureWeightsGradient, featureBiasesGradient, outputWeightsGradient
        );
        hipDeviceSynchronize();

        backpropOutputBias<<<1, 1>>>(batchSize, outputs, outputBiasesGradient);
        hipDeviceSynchronize();

        hipFree(ourAccumulators);
        hipFree(oppAccumulators);
        hipFree(outputs);
        hipDeviceSynchronize();

        return hipGetLastError();
    }
}