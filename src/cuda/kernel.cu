#pragma once
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdint>

__global__ void addInternal(const float* A, const float* B, float* C, int size)
{
    int i = threadIdx.x;

    if (i >= size)
        return;

    C[i] = A[i] + B[i];
}

__global__ void populateAccumulator(
    const size_t batchSize,
    const size_t hiddenSize,
    const size_t inputSize,
    const float* featureWeights,
    const float* featureBiases,
    const uint16_t* inputs,
    float* accumulators)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= hiddenSize)
        return;

    const size_t inputIdx = inputSize * blockIdx.x;
    const size_t element = threadIdx.x;
    const size_t outputIdx = hiddenSize * blockIdx.x + element;

    const uint16_t* thisInput = inputs + inputIdx;

    float elementVal = featureBiases[element];

    for (int i = 0; i < inputSize; i++) {
        if (thisInput[i] >= 768)
            break;

        const size_t idx = thisInput[i] * hiddenSize + element;
        elementVal += featureWeights[idx];
    }

    elementVal = elementVal < 0 ? 0 : elementVal > 1 ? 1 : elementVal;

    accumulators[outputIdx] = elementVal;
}

__global__ void setOutputBias(
    const size_t batchSize,
    const float* outputBias,
    float* outputs)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= batchSize)
        return;

    outputs[idx] = outputBias[0];
}

__global__ void calculateEvals(
    const size_t batchSize,
    const size_t hiddenSize,
    const float* outputWeights,
    const float* outputBiases,
    const float* ourAccumulators,
    const float* oppAccumulators,
    float* outputs)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= hiddenSize)
        return;

    const size_t element = threadIdx.x;
    const size_t outputIdx = blockIdx.x;
    const size_t idx = outputIdx * hiddenSize + element;

    float outputVal = ourAccumulators[idx] * outputWeights[element];
    outputVal += oppAccumulators[idx] * outputWeights[hiddenSize + element];

    atomicAdd(&outputs[outputIdx], outputVal);
}

__global__ void calculateErrors(
    const size_t batchSize,
    const size_t hiddenSize,
    const float* results,
    float* outputs,
    float* error)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= batchSize)
        return;

    const float eval = outputs[idx];
    const float result = results[idx];
    const float sigmoid = 1.0 / (1.0 + expf(-eval));
    const float diff = sigmoid - result;
    const float singleError = diff * sigmoid * (1.0 - sigmoid);

    atomicAdd(error, diff * diff);

    outputs[idx] = singleError;
}

__global__ void backpropSide(
    const size_t batchSize,
    const size_t hiddenSize,
    const size_t inputSize,
    const size_t outputOffset,
    const float* outputWeights,
    const float* accumulator,
    const uint16_t* inputs,
    const float* outputs,
    float* featureWeightsGradient,
    float* featureBiasesGradient,
    float* outputWeightsGradient)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= hiddenSize)
        return;

    const size_t element = threadIdx.x;
    const size_t outputIdx = blockIdx.x;
    const size_t inputIdx = outputIdx * inputSize;
    const size_t outputWeightIdx = element + outputOffset;
    const size_t accumulatorIdx = outputIdx * hiddenSize + element;

    const uint16_t* thisInput = inputs + inputIdx;

    const float error = outputs[outputIdx];
    const float weight = outputWeights[outputWeightIdx];
    const float accumulatorVal = accumulator[accumulatorIdx];

    // uses a trick
    const float component = accumulatorVal > 0 && accumulatorVal < 1
        ? error * weight
        : 0;

    atomicAdd(&featureBiasesGradient[element], component);
    atomicAdd(&outputWeightsGradient[outputWeightIdx], error * accumulatorVal);

    for (int i = 0; i < inputSize; i++) {
        if (thisInput[i] >= 768)
            break;

        const size_t x = thisInput[i] * hiddenSize + element;
        atomicAdd(&featureWeightsGradient[x], component);
    }
}

__global__ void backpropOutputBias(
    const size_t batchSize,
    const float* outputs,
    float* outputBiasesGradient)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= batchSize)
        return;

    atomicAdd(outputBiasesGradient, outputs[idx]);
}

void checkError(std::string message)
{
    const auto error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cout << message << std::endl;
        std::cout << hipGetErrorString(error) << std::endl;
    }
}

extern "C" {
    hipError_t add(const float* A, const float* B, float* C, int size)
    {
        addInternal<<<1, 3>>>(A, B, C, size);

        return hipGetLastError();
    }

    hipError_t trainBatch(
        const size_t batchSize,
        const size_t hiddenSize,
        const size_t inputSize,
        const float* featureWeights,
        const float* featureBiases,
        const float* outputWeights,
        const float* outputBiases,
        const uint16_t* ourInputs,
        const uint16_t* oppInputs,
        const float* results,
        float* featureWeightsGradient,
        float* featureBiasesGradient,
        float* outputWeightsGradient,
        float* outputBiasesGradient,
        float* error)
    {
        const size_t accumulatorSize = batchSize * hiddenSize * sizeof(float);
        const size_t outputSize = batchSize * sizeof(float);
        const size_t blocks = (batchSize + hiddenSize - 1) / hiddenSize;

        float* ourAccumulators;
        hipMalloc(&ourAccumulators, accumulatorSize);
        hipDeviceSynchronize();
        checkError("malloc 1");

        populateAccumulator<<<batchSize, hiddenSize>>>(batchSize, hiddenSize, inputSize, featureWeights, featureBiases, ourInputs, ourAccumulators);
        hipDeviceSynchronize();
        checkError("accumulator 1");

        float* oppAccumulators;
        hipMalloc(&oppAccumulators, accumulatorSize);
        hipDeviceSynchronize();
        checkError("malloc 2");

        populateAccumulator<<<batchSize, hiddenSize>>>(batchSize, hiddenSize, inputSize, featureWeights, featureBiases, oppInputs, oppAccumulators);
        hipDeviceSynchronize();
        checkError("accumulator 2 ");

        float* outputs;
        hipMallocManaged(&outputs, outputSize);
        hipDeviceSynchronize();
        checkError("malloc 3");

        setOutputBias<<<blocks, hiddenSize>>>(batchSize, outputBiases, outputs);
        hipDeviceSynchronize();
        checkError("memset");

        calculateEvals<<<batchSize, hiddenSize>>>(batchSize, hiddenSize, outputWeights, outputBiases, ourAccumulators, oppAccumulators, outputs);
        hipDeviceSynchronize();
        checkError("eval");

        calculateErrors<<<blocks, hiddenSize>>>(batchSize, hiddenSize, results, outputs, error);
        hipDeviceSynchronize();
        checkError("error");

        backpropSide<<<batchSize, hiddenSize>>>(
            batchSize, hiddenSize, inputSize, 0,
            outputWeights, ourAccumulators, ourInputs, outputs,
            featureWeightsGradient, featureBiasesGradient, outputWeightsGradient
        );
        hipDeviceSynchronize();
        checkError("backprops 1");

        backpropSide<<<batchSize, hiddenSize>>>(
            batchSize, hiddenSize, inputSize, hiddenSize,
            outputWeights, oppAccumulators, oppInputs, outputs,
            featureWeightsGradient, featureBiasesGradient, outputWeightsGradient
        );
        hipDeviceSynchronize();
        checkError("backprops 2");

        backpropOutputBias<<<1, 1>>>(batchSize, outputs, outputBiasesGradient);
        hipDeviceSynchronize();
        checkError("backprops 3");

        hipFree(ourAccumulators);
        hipFree(oppAccumulators);
        hipFree(outputs);
        hipDeviceSynchronize();
        checkError("freeing");

        return hipGetLastError();
    }
}