#pragma once
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdint>

#define INPUT 32
#define HIDDEN 768

__global__ void addInternal(const float* A, const float* B, float* C, int size)
{
    int i = threadIdx.x;

    if (i >= size)
        return;

    C[i] = A[i] + B[i];
}

__global__ void populateAccumulator(
    const size_t batchSize,
    const float* featureWeights,
    const float* featureBiases,
    const uint16_t* inputs,
    float* accumulators)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= HIDDEN)
        return;

    const size_t inputIdx = INPUT * blockIdx.x;
    const size_t element = threadIdx.x;
    const size_t outputIdx = HIDDEN * blockIdx.x + element;

    const uint16_t* thisInput = inputs + inputIdx;

    float elementVal = featureBiases[element];

    for (size_t i = 0; i < INPUT; i++) {
        if (thisInput[i] >= static_cast<uint16_t>(768))
            break;

        const size_t idx = static_cast<size_t>(thisInput[i]) * HIDDEN + element;
        elementVal += featureWeights[idx];
    }

    if (elementVal < 0)
        elementVal = 0;
    else if (elementVal > 1)
        elementVal = 1;

    accumulators[outputIdx] = elementVal;
}

__global__ void calculateErrors(
    const size_t batchSize,
    const float* outputWeights,
    const float* outputBiases,
    const float* ourAccumulators,
    const float* oppAccumulators,
    const float* results,
    float* outputs,
    float* error)
{
    const size_t outputIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (outputIdx >= batchSize)
        return;

    const size_t accumulatorIdx = outputIdx * HIDDEN;

    float eval = outputBiases[0];

    for (size_t i = 0; i < HIDDEN; i++)
        eval += ourAccumulators[accumulatorIdx + i] * outputWeights[i];;

    for (size_t i = 0; i < HIDDEN; i++)
        eval += oppAccumulators[accumulatorIdx + i] * outputWeights[HIDDEN + i];

    const float sigmoid = 1.0 / (1.0 + expf(-eval));
    const float diff = sigmoid - results[outputIdx];
    const float singleError = diff * sigmoid * (1.0 - sigmoid);

    atomicAdd(error, diff * diff);

    outputs[outputIdx] = singleError;
}

__global__ void backpropSide(
    const size_t batchSize,
    const size_t outputOffset,
    const float* outputWeights,
    const float* accumulator,
    const uint16_t* inputs,
    const float* outputs,
    float* featureWeightsGradient,
    float* featureBiasesGradient,
    float* outputWeightsGradient)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= HIDDEN)
        return;

    const size_t element = threadIdx.x;
    const size_t outputIdx = blockIdx.x;
    const size_t inputIdx = outputIdx * INPUT;
    const size_t outputWeightIdx = element + outputOffset;
    const size_t accumulatorIdx = outputIdx * HIDDEN + element;

    const uint16_t* thisInput = inputs + inputIdx;

    const float error = outputs[outputIdx];
    const float weight = outputWeights[outputWeightIdx];
    const float accumulatorVal = accumulator[accumulatorIdx];

    // uses a trick
    const float component = accumulatorVal > 0 && accumulatorVal < 1
        ? error * weight
        : 0;

    atomicAdd(&featureBiasesGradient[element], component);
    atomicAdd(&outputWeightsGradient[outputWeightIdx], error * accumulatorVal);

    for (int i = 0; i < INPUT; i++) {
        if (thisInput[i] >= static_cast<uint16_t>(768))
            break;

        const size_t x = thisInput[i] * HIDDEN + element;
        atomicAdd(&featureWeightsGradient[x], component);
    }
}

__global__ void backpropOutputBias(
    const size_t batchSize,
    const float* outputs,
    float* outputBiasesGradient)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= batchSize)
        return;

    atomicAdd(outputBiasesGradient, outputs[idx]);
}

void checkError(std::string message)
{
    const auto error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cout << message << std::endl;
        std::cout << hipGetErrorString(error) << std::endl;
    }
}

size_t calcBlocks(size_t total, size_t threads)
{
    return (total + threads - 1) / threads;
}

extern "C" {
    hipError_t add(const float* A, const float* B, float* C, int size)
    {
        addInternal<<<1, 3>>>(A, B, C, size);

        return hipGetLastError();
    }

    hipError_t trainBatch(
        const size_t batchSize,
        const size_t hiddenSize,
        const size_t inputSize,
        const float* featureWeights,
        const float* featureBiases,
        const float* outputWeights,
        const float* outputBiases,
        const uint16_t* ourInputs,
        const uint16_t* oppInputs,
        const float* results,
        float* featureWeightsGradient,
        float* featureBiasesGradient,
        float* outputWeightsGradient,
        float* outputBiasesGradient,
        float* error)
    {
        if (inputSize != INPUT)
        {
            std::cout << "Incompatible input format.";
            exit(1);
        }

        if (hiddenSize != HIDDEN)
        {
            std::cout << "HIDDEN must be set to " << hiddenSize << " in src/cuda/kernel.cu";
            exit(1);
        }

        const size_t accumulatorSize = batchSize * HIDDEN * sizeof(float);
        const size_t outputSize = batchSize * sizeof(float);
        const size_t blocks = calcBlocks(batchSize, HIDDEN);
        const size_t sumBlocks = calcBlocks(batchSize, 1024);

        float* ourAccumulators;
        hipMalloc(&ourAccumulators, accumulatorSize);
        float* oppAccumulators;
        hipMalloc(&oppAccumulators, accumulatorSize);
        hipDeviceSynchronize();

        populateAccumulator<<<batchSize, HIDDEN>>>(batchSize, featureWeights, featureBiases, ourInputs, ourAccumulators);
        hipDeviceSynchronize();

        populateAccumulator<<<batchSize, HIDDEN>>>(batchSize, featureWeights, featureBiases, oppInputs, oppAccumulators);
        hipDeviceSynchronize();

        float* outputs;
        hipMalloc(&outputs, outputSize);
        hipDeviceSynchronize();

        calculateErrors<<<sumBlocks, 1024>>>(batchSize, outputWeights, outputBiases, ourAccumulators, oppAccumulators, results, outputs, error);
        hipDeviceSynchronize();

        backpropSide<<<batchSize, HIDDEN>>>(
            batchSize, 0,
            outputWeights, ourAccumulators, ourInputs, outputs,
            featureWeightsGradient, featureBiasesGradient, outputWeightsGradient
        );
        hipDeviceSynchronize();

        backpropSide<<<batchSize, HIDDEN>>>(
            batchSize, HIDDEN,
            outputWeights, oppAccumulators, oppInputs, outputs,
            featureWeightsGradient, featureBiasesGradient, outputWeightsGradient
        );
        hipDeviceSynchronize();

        backpropOutputBias<<<sumBlocks, 1024>>>(batchSize, outputs, outputBiasesGradient);
        hipDeviceSynchronize();

        hipFree(ourAccumulators);
        hipFree(oppAccumulators);
        hipFree(outputs);
        hipDeviceSynchronize();

        return hipGetLastError();
    }
}