
#include <hip/hip_runtime.h>
#ifdef __HIP_PLATFORM_AMD__
#include <hip/hip_runtime.h>
#endif

constexpr size_t threadsPerBlock = static_cast<size_t>(1024);

__global__ void powerErrorKernel(
    const size_t bufferSize,
    const float* inputs,
    const float* results,
    float* output,
    const float power)
{
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= bufferSize)
        return;

    const float absd = abs(inputs[i] - results[i]);
    const float error = powf(absd, power);

    atomicAdd(output, error);
}

__global__ void backpropPowerErrorKernel(
    const size_t bufferSize,
    const float* inputs,
    const float* results,
    const float* output_grad,
    float* input_grads,
    const float power)
{
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= bufferSize)
        return;

    const float diff = inputs[i] - results[i];
    const float absd = abs(diff);

    const float grad = power * powf(absd, power - 1) * (*output_grad);
    input_grads[i] = diff > 0.0F ? grad : -grad;
}

extern "C" void powerError(
    const size_t bufferSize,
    const float* inputs,
    const float* results,
    float* output,
    const float power)
{
    const size_t numBlocks = (bufferSize + threadsPerBlock - 1) / threadsPerBlock;
    powerErrorKernel<<<numBlocks, threadsPerBlock>>>(bufferSize, inputs, results, output, power);
}

extern "C" void backpropPowerError(
    const size_t bufferSize,
    const float* inputs,
    const float* results,
    const float* output_grad,
    float* input_grads,
    const float power)
{
    const size_t numBlocks = (bufferSize + threadsPerBlock - 1) / threadsPerBlock;
    backpropPowerErrorKernel<<<numBlocks, threadsPerBlock>>>(bufferSize, inputs, results, output_grad, input_grads, power);
}
