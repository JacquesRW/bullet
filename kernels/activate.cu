#include "hip/hip_runtime.h"
#include "util.cu"
#ifdef __HIP_PLATFORM_AMD__
#include <hip/hip_runtime.h>
#endif

typedef float(*OpType)(float);

template<OpType op>
__global__ void bufferOperation(const size_t size, const float* in, float* out)
{
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= size)
        return;

    out[i] = op(in[i]);
}

template<OpType op>
__global__ void bufferBackprop(const size_t size, const float* input, const float* output_grad, float* input_grad)
{
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= size)
        return;

    const float thisIn = input[i];
    const float thisOutGrd = output_grad[i];

    input_grad[i] = op(thisIn) * thisOutGrd;
}

extern "C" void backpropReLU(const size_t size, const float* input, const float* output_grad, float* input_grad)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    bufferBackprop<primeReLU><<<numBlocks, threadsPerBlock>>>(size, input, output_grad, input_grad);
}

extern "C" void backpropCReLU(const size_t size, const float* input, const float* output_grad, float* input_grad)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    bufferBackprop<primeCReLU><<<numBlocks, threadsPerBlock>>>(size, input, output_grad, input_grad);
}

extern "C" void backpropSCReLU(const size_t size, const float* input, const float* output_grad, float* input_grad)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    bufferBackprop<primeSCReLU><<<numBlocks, threadsPerBlock>>>(size, input, output_grad, input_grad);
}

extern "C" void backpropSqrReLU(const size_t size, const float* input, const float* output_grad, float* input_grad)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    bufferBackprop<primeSqrReLU><<<numBlocks, threadsPerBlock>>>(size, input, output_grad, input_grad);
}

extern "C" void backpropSigmoid(const size_t size, const float* output, const float* output_grad, float* input_grad)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    bufferBackprop<primeSqrReLU><<<numBlocks, threadsPerBlock>>>(size, output, output_grad, input_grad);
}

extern "C" void activateReLU(const size_t size, const float* in, float* out)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    bufferOperation<ReLU><<<numBlocks, threadsPerBlock>>>(size, in, out);
}

extern "C" void activateCReLU(const size_t size, const float* in, float* out)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    bufferOperation<CReLU><<<numBlocks, threadsPerBlock>>>(size, in, out);
}

extern "C" void activateSCReLU(const size_t size, const float* in, float* out)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    bufferOperation<SCReLU><<<numBlocks, threadsPerBlock>>>(size, in, out);
}

extern "C" void activateSqrReLU(const size_t size, const float* in, float* out)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    bufferOperation<SqrReLU><<<numBlocks, threadsPerBlock>>>(size, in, out);
}

extern "C" void activateSigmoid(const size_t size, const float* in, float* out)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    bufferOperation<sigmoid><<<numBlocks, threadsPerBlock>>>(size, in, out);
}
