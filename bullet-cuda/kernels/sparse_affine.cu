#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdint>

struct Feat {
    uint16_t our;
    uint16_t opp;
};

__global__ void __kernel_sparse_affine_forward(
    const size_t inputSize,
    const size_t outputSize,
    const float* weights,
    const float* biases,
    const Feat* inputs,
    float* outputs)
{
    const size_t elem = blockIdx.x * blockDim.x + threadIdx.x;

    if (elem >= outputSize)
        return;

    const size_t inputIdx = inputSize * blockIdx.y;
    const Feat* thisInput = inputs + inputSize * blockIdx.y;
    float* thisOutput = outputs + 2 * outputSize * blockIdx.y + elem;

    float ourElementVal = biases[elem];
    float oppElementVal = ourElementVal;

    for (size_t i = 0; i < inputSize; i++) {
        const Feat inp = thisInput[i];

        if (static_cast<size_t>(inp.our) == static_cast<size_t>(65535))
            break;

        const size_t ourIdx = static_cast<size_t>(inp.our) * outputSize + elem;
        const size_t oppIdx = static_cast<size_t>(inp.opp) * outputSize + elem;
        ourElementVal += weights[ourIdx];
        oppElementVal += weights[oppIdx];
    }

    thisOutput[         0] = ourElementVal;
    thisOutput[outputSize] = oppElementVal;
}

__global__ void __kernel_sparse_affine_backward(
    const size_t inputSize,
    const size_t outputSize,
    float* weightsGrad,
    float* biasesGrad,
    const Feat* inputs,
    const float* errors)
{
    const size_t elem = blockIdx.x * blockDim.x + threadIdx.x;

    if (elem >= outputSize)
        return;

    const Feat* thisInput = inputs + inputSize * blockIdx.y;
    const float* thisErrors = errors + 2 * outputSize * blockIdx.y;

    const float ourError = thisErrors[elem];
    const float oppError = thisErrors[elem + outputSize];

    atomicAdd(&biasesGrad[elem], ourError + oppError);

    for (size_t i = 0; i < inputSize; i++) {
        const Feat inp = thisInput[i];

        if (static_cast<size_t>(inp.our) == static_cast<size_t>(65535))
            break;

        const size_t ourIdx = static_cast<size_t>(inp.our) * outputSize + elem;
        const size_t oppIdx = static_cast<size_t>(inp.opp) * outputSize + elem;
        atomicAdd(&weightsGrad[ourIdx], ourError);
        atomicAdd(&weightsGrad[oppIdx], oppError);
    }
}

extern "C" void sparseAffineForward(
    const size_t batchSize,
    const size_t maxInputSize,
    const size_t outputSize,
    const float* weights,
    const float* biases,
    const Feat* inputs,
    float* outputs)
{
    const size_t numChunks = static_cast<size_t>(1) + outputSize / static_cast<size_t>(1024);

    dim3 grid(numChunks, batchSize);

    const size_t threads = (numChunks == 1) ? outputSize : 1024;

    __kernel_sparse_affine_forward<<<grid, threads>>>(
        maxInputSize,
        outputSize,
        weights,
        biases,
        inputs,
        outputs
    );
}

extern "C" void sparseAffineBackward(
    const size_t batchSize,
    const size_t maxInputSize,
    const size_t outputSize,
    float* weightsGrad,
    float* biasesGrad,
    const Feat* inputs,
    const float* errors)
{
    const size_t numChunks = static_cast<size_t>(1) + outputSize / static_cast<size_t>(1024);

    dim3 grid(numChunks, batchSize);

    const size_t threads = (numChunks == 1) ? outputSize : 1024;

    __kernel_sparse_affine_backward<<<grid, threads>>>(
        maxInputSize,
        outputSize,
        weightsGrad,
        biasesGrad,
        inputs,
        errors
    );
}
