#include "hip/hip_runtime.h"
/*
Calculating the gradient for a batch.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>

#include "util.h"

__global__ void populateAccumulator(
    const size_t batchSize,
    const float* featureWeights,
    const float* featureBiases,
    const uint16_t* inputs,
    float* accumulators)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= HIDDEN)
        return;

    const size_t inputIdx = INPUT * blockIdx.x;
    const size_t element = threadIdx.x;
    const size_t outputIdx = HIDDEN * blockIdx.x + element;

    const uint16_t* thisInput = inputs + inputIdx;

    float elementVal = featureBiases[element];

    for (size_t i = 0; i < INPUT; i++) {
        if (thisInput[i] == static_cast<uint16_t>(65535))
            break;

        const size_t idx = static_cast<size_t>(thisInput[i]) * HIDDEN + element;
        elementVal += featureWeights[idx];
    }

    elementVal = activate(elementVal);

    accumulators[outputIdx] = elementVal;
}

__global__ void calculateErrors(
    const size_t batchSize,
    const float* outputWeights,
    const float* outputBiases,
    const float* ourAccumulators,
    const float* oppAccumulators,
    const float* results,
    float* outputs,
    float* error)
{
    const size_t outputIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (outputIdx >= batchSize)
        return;

    const size_t accumulatorIdx = outputIdx * HIDDEN;

    float eval = outputBiases[0];

    for (size_t i = 0; i < HIDDEN; i++)
        eval += ourAccumulators[accumulatorIdx + i] * outputWeights[i];;

    for (size_t i = 0; i < HIDDEN; i++)
        eval += oppAccumulators[accumulatorIdx + i] * outputWeights[HIDDEN + i];

    const float sigmoid = 1.0 / (1.0 + expf(-eval));
    const float diff = sigmoid - results[outputIdx];
    const float singleError = diff * sigmoid * (1.0 - sigmoid);

    atomicAdd(error, diff * diff);

    outputs[outputIdx] = singleError;
}

__global__ void backpropSide(
    const size_t batchSize,
    const size_t outputOffset,
    const float* outputWeights,
    const float* accumulator,
    const uint16_t* inputs,
    const float* outputs,
    float* featureWeightsGradient,
    float* featureBiasesGradient,
    float* outputWeightsGradient)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= HIDDEN)
        return;

    const size_t element = threadIdx.x;
    const size_t outputIdx = blockIdx.x;
    const size_t inputIdx = outputIdx * INPUT;
    const size_t outputWeightIdx = element + outputOffset;
    const size_t accumulatorIdx = outputIdx * HIDDEN + element;

    const uint16_t* thisInput = inputs + inputIdx;

    const float error = outputs[outputIdx];
    const float weight = outputWeights[outputWeightIdx];
    const float accumulatorVal = accumulator[accumulatorIdx];

    // uses a trick
    const float component = prime(accumulatorVal) * error * weight;

    atomicAdd(&featureBiasesGradient[element], component);
    atomicAdd(&outputWeightsGradient[outputWeightIdx], error * accumulatorVal);

    for (int i = 0; i < INPUT; i++) {
        if (thisInput[i] == static_cast<uint16_t>(65535))
            break;

        const size_t x = thisInput[i] * HIDDEN + element;
        atomicAdd(&featureWeightsGradient[x], component);
    }
}

__global__ void backpropOutputBias(
    const size_t batchSize,
    const float* outputs,
    float* outputBiasesGradient)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= batchSize)
        return;

    atomicAdd(outputBiasesGradient, outputs[idx]);
}

extern "C" hipError_t calcGradient(
    const size_t batchSize,
    const size_t hiddenSize,
    const size_t inputSize,
    const float* featureWeights,
    const float* featureBiases,
    const float* outputWeights,
    const float* outputBiases,
    const uint16_t* ourInputs,
    const uint16_t* oppInputs,
    const float* results,
    float* featureWeightsGradient,
    float* featureBiasesGradient,
    float* outputWeightsGradient,
    float* outputBiasesGradient,
    float* error,
    float* ourAccumulators,
    float* oppAccumulators,
    float* outputs)
{
    if (inputSize != INPUT)
    {
        std::cout << "Incompatible input format.";
        exit(1);
    }

    if (hiddenSize != HIDDEN)
    {
        std::cout << "HIDDEN must be set to " << hiddenSize << " in src/cuda/kernel.cu";
        exit(1);
    }

    const size_t blocks = calcBlocks(batchSize, HIDDEN);
    const size_t sumBlocks = calcBlocks(batchSize, 1024);

    populateAccumulator<<<batchSize, HIDDEN>>>(batchSize, featureWeights, featureBiases, ourInputs, ourAccumulators);

    populateAccumulator<<<batchSize, HIDDEN>>>(batchSize, featureWeights, featureBiases, oppInputs, oppAccumulators);

    calculateErrors<<<sumBlocks, 1024>>>(batchSize, outputWeights, outputBiases, ourAccumulators, oppAccumulators, results, outputs, error);

    backpropSide<<<batchSize, HIDDEN>>>(
        batchSize, 0,
        outputWeights, ourAccumulators, ourInputs, outputs,
        featureWeightsGradient, featureBiasesGradient, outputWeightsGradient
    );

    backpropSide<<<batchSize, HIDDEN>>>(
        batchSize, HIDDEN,
        outputWeights, oppAccumulators, oppInputs, outputs,
        featureWeightsGradient, featureBiasesGradient, outputWeightsGradient
    );

    backpropOutputBias<<<sumBlocks, 1024>>>(batchSize, outputs, outputBiasesGradient);

    hipDeviceSynchronize();

    return hipGetLastError();
}
