#pragma once
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdint>

#ifndef HIDDEN
#define HIDDEN 768
#endif

#ifndef INPUT
#define INPUT 32
#endif

#if defined(RELU)
    __device__ float activate(float in) { return in > 0 ? in : 0; }
    __device__ float prime(float in) { return in > 0 ? 1 : 0; }
#elif defined(SCRELU)
    __device__ float activate(float in) { return in < 0 ? 0 : (in > 1 ? 1 : (in * in)); }
    __device__ float prime(float in) { return in > 0 && in < 1 ? 2 * in : 0; }
#elif defined(FASTSCRELU)
    constexpr float fastFactor = 255.0 / 256.0;
    __device__ float activate(float in)
    {
        const float sq = in * in * fastFactor;
        return sq < 0 ? 0 : (sq > 1 ? 1 : sq);
    }
    __device__ float prime(float in) { return fastFactor * (in > 0 && in < 1 ? 2 * in : 0); }
#elif defined(CRELU)
    __device__ float activate(float in) { return in < 0 ? 0 : (in > 1 ? 1 : in); }
    __device__ float prime(float in) { return in > 0 && in < 1 ? 1 : 0; }
#else
    __device__ float activate(float in);
    __device__ float prime(float in);
#endif

__global__ void populateAccumulator(
    const size_t batchSize,
    const float* featureWeights,
    const float* featureBiases,
    const uint16_t* inputs,
    float* accumulators)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= HIDDEN)
        return;

    const size_t inputIdx = INPUT * blockIdx.x;
    const size_t element = threadIdx.x;
    const size_t outputIdx = HIDDEN * blockIdx.x + element;

    const uint16_t* thisInput = inputs + inputIdx;

    float elementVal = featureBiases[element];

    for (size_t i = 0; i < INPUT; i++) {
        if (thisInput[i] == static_cast<uint16_t>(65535))
            break;

        const size_t idx = static_cast<size_t>(thisInput[i]) * HIDDEN + element;
        elementVal += featureWeights[idx];
    }

    elementVal = activate(elementVal);

    accumulators[outputIdx] = elementVal;
}

__global__ void calculateErrors(
    const size_t batchSize,
    const float* outputWeights,
    const float* outputBiases,
    const float* ourAccumulators,
    const float* oppAccumulators,
    const float* results,
    float* outputs,
    float* error)
{
    const size_t outputIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (outputIdx >= batchSize)
        return;

    const size_t accumulatorIdx = outputIdx * HIDDEN;

    float eval = outputBiases[0];

    for (size_t i = 0; i < HIDDEN; i++)
        eval += ourAccumulators[accumulatorIdx + i] * outputWeights[i];;

    for (size_t i = 0; i < HIDDEN; i++)
        eval += oppAccumulators[accumulatorIdx + i] * outputWeights[HIDDEN + i];

    const float sigmoid = 1.0 / (1.0 + expf(-eval));
    const float diff = sigmoid - results[outputIdx];
    const float singleError = diff * sigmoid * (1.0 - sigmoid);

    atomicAdd(error, diff * diff);

    outputs[outputIdx] = singleError;
}

__global__ void backpropSide(
    const size_t batchSize,
    const size_t outputOffset,
    const float* outputWeights,
    const float* accumulator,
    const uint16_t* inputs,
    const float* outputs,
    float* featureWeightsGradient,
    float* featureBiasesGradient,
    float* outputWeightsGradient)
{
    if (blockIdx.x >= batchSize)
        return;

    if (threadIdx.x >= HIDDEN)
        return;

    const size_t element = threadIdx.x;
    const size_t outputIdx = blockIdx.x;
    const size_t inputIdx = outputIdx * INPUT;
    const size_t outputWeightIdx = element + outputOffset;
    const size_t accumulatorIdx = outputIdx * HIDDEN + element;

    const uint16_t* thisInput = inputs + inputIdx;

    const float error = outputs[outputIdx];
    const float weight = outputWeights[outputWeightIdx];
    const float accumulatorVal = accumulator[accumulatorIdx];

    // uses a trick
    const float component = prime(accumulatorVal) * error * weight;

    atomicAdd(&featureBiasesGradient[element], component);
    atomicAdd(&outputWeightsGradient[outputWeightIdx], error * accumulatorVal);

    for (int i = 0; i < INPUT; i++) {
        if (thisInput[i] == static_cast<uint16_t>(65535))
            break;

        const size_t x = thisInput[i] * HIDDEN + element;
        atomicAdd(&featureWeightsGradient[x], component);
    }
}

__global__ void backpropOutputBias(
    const size_t batchSize,
    const float* outputs,
    float* outputBiasesGradient)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= batchSize)
        return;

    atomicAdd(outputBiasesGradient, outputs[idx]);
}

void checkError(std::string message)
{
    const auto error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cout << message << std::endl;
        std::cout << hipGetErrorString(error) << std::endl;
    }
}

size_t calcBlocks(size_t total, size_t threads)
{
    return (total + threads - 1) / threads;
}

extern "C" hipError_t calcGradient(
    const size_t batchSize,
    const size_t hiddenSize,
    const size_t inputSize,
    const float* featureWeights,
    const float* featureBiases,
    const float* outputWeights,
    const float* outputBiases,
    const uint16_t* ourInputs,
    const uint16_t* oppInputs,
    const float* results,
    float* featureWeightsGradient,
    float* featureBiasesGradient,
    float* outputWeightsGradient,
    float* outputBiasesGradient,
    float* error,
    float* ourAccumulators,
    float* oppAccumulators,
    float* outputs)
{
    if (inputSize != INPUT)
    {
        std::cout << "Incompatible input format.";
        exit(1);
    }

    if (hiddenSize != HIDDEN)
    {
        std::cout << "HIDDEN must be set to " << hiddenSize << " in src/cuda/kernel.cu";
        exit(1);
    }

    const size_t blocks = calcBlocks(batchSize, HIDDEN);
    const size_t sumBlocks = calcBlocks(batchSize, 1024);

    populateAccumulator<<<batchSize, HIDDEN>>>(batchSize, featureWeights, featureBiases, ourInputs, ourAccumulators);

    populateAccumulator<<<batchSize, HIDDEN>>>(batchSize, featureWeights, featureBiases, oppInputs, oppAccumulators);

    calculateErrors<<<sumBlocks, 1024>>>(batchSize, outputWeights, outputBiases, ourAccumulators, oppAccumulators, results, outputs, error);

    backpropSide<<<batchSize, HIDDEN>>>(
        batchSize, 0,
        outputWeights, ourAccumulators, ourInputs, outputs,
        featureWeightsGradient, featureBiasesGradient, outputWeightsGradient
    );

    backpropSide<<<batchSize, HIDDEN>>>(
        batchSize, HIDDEN,
        outputWeights, oppAccumulators, oppInputs, outputs,
        featureWeightsGradient, featureBiasesGradient, outputWeightsGradient
    );

    backpropOutputBias<<<sumBlocks, 1024>>>(batchSize, outputs, outputBiasesGradient);

    hipDeviceSynchronize();

    return hipGetLastError();
}
